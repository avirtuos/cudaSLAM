#include "hip/hip_runtime.h"

#include "Map.h"

Map::Map(int width_arg, int height_arg, int scan_buffer_size)
{
    map_init = false;
    width = width_arg;
    height = height_arg;

    //Allocate scan buffer on the device as well as the size of scan
    const unsigned int scan_bytes = scan_buffer_size * sizeof(TelemetryPoint);
    checkCuda(hipMalloc((void **)&scan_buffer_d, scan_bytes));
    checkCuda(hipMalloc((void **)&scan_size_d, sizeof(int)));

    //Allocate pinned memory on the host and device for the current map
    map_bytes = width * height * sizeof(MapPoint);
    checkCuda(hipHostMalloc((void**)&map_h, map_bytes));
    checkCuda(hipMalloc((void**)&map_d, map_bytes));
    checkCuda(hipMalloc((void **)&width_d, sizeof(int)));
    checkCuda(hipMalloc((void **)&height_d, sizeof(int)));

    //copy the size of the map to the device
    //todo: in the future this should probably be auto-expandable based on the size of the mapped area
    //but evne using a static size is fine for area 30m x 30m which is more than enough for most hobby applications
    hipMemcpy(width_d, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(height_d, &height, sizeof(int), hipMemcpyHostToDevice);
}

Map::~Map()
{
    hipHostFree(map_h);
    hipFree(map_d);
    hipFree(width_d);
    hipFree(height_d);
    hipFree(scan_buffer_d);
    hipFree(scan_size_d);
}

//TODO: There is opportunity to speed this up using hints from odometry or even just simple distance traveled estimates.
__global__
void cudaUpdateMap(int n, LocalizedOrigin *result, TelemetryPoint *scan_buffer, int *scan_size, MapPoint *map, int* map_width, int* map_height)
{
    int offset = blockIdx.x*blockDim.x + threadIdx.x;
    if(offset > 10) {
        return;
    }

    int x_offset = -512 + (int)threadIdx.x;
    int y_offset = -512 + (int)blockIdx.x;

    LocalizedOrigin best;
    best.score = -1;

    //Try various angles - TODO: find better sampling technique here possibly even re-sampling
    for(int angle_offset = 0; angle_offset < 360; angle_offset++){
        //For each point see if we have a hit

        LocalizedOrigin current_sim;
        current_sim.x_offset = x_offset;
        current_sim.y_offset = y_offset;
        current_sim.angle_offset = angle_offset;
        current_sim.score = 0;

        for(int scan_point = 0; scan_point < *scan_size; scan_point++){
            int angle_offset = 0;

            float distance = scan_buffer[scan_point].distance;
            float angle_radians = scan_buffer[scan_point].angle;

            int x = x_offset + roundf(sin (angle_offset + (angle_radians * 3.14159265 / 180)) * distance);
            int y = y_offset + roundf(cos (angle_offset + (angle_radians * 3.14159265 / 180)) * distance);

            int pos = ((*map_height/2 + y) * *map_width) + (*map_width/2 + x);
            MapPoint *map_point = map+pos;
            if(map_point->occupancy > 0){
                current_sim.score++;
            }
        }

        if(best.score < current_sim.score) {
            best.x_offset = current_sim.x_offset;
            best.y_offset = current_sim.y_offset;
            best.angle_offset = current_sim.angle_offset;
            best.score = current_sim.score;
        }
    }


    result[offset].x_offset = best.x_offset;
    result[offset].y_offset = best.y_offset;
    result[offset].angle_offset = best.angle_offset;
    result[offset].score = best.score;
    printf("Particle Filter Result: i[%d] x[%d] y[%d] angle[%.2f] score[%d]\n", offset, best.x_offset, best.y_offset, best.angle_offset, best.score);
}


//TODO: So far we've only been working on Localization, we need to start thinking about mapping or rather
//when to update the map with newly scanned points. I suspect that cold start might be a special case but 
//it needs more thinking. I like the idea of the map being fully mutable, not just additive which is what
//I've seen from other SLAM impls.
__global__
void cudaInitMap(TelemetryPoint *scan_buffer, int *scan_size, MapPoint *map, int* map_width, int* map_height)
{
    for(int i = 0; i < *scan_size; i++){
        TelemetryPoint *cur_point = scan_buffer+i;
        int pos = ((*map_height/2 + cur_point->y) * *map_width) + (*map_height/2 + cur_point->x);
        MapPoint *cur_map = map+pos;
        if(cur_map->occupancy < 1000) {
            cur_map->occupancy++;
        }
    }
}


TelemetryPoint Map::update(int32_t search_distance, TelemetryPoint scan_data[], int scan_size)
{
    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );
    checkCuda( hipEventRecord(startEvent, 0) );

    const unsigned int bytes = scan_size * sizeof(TelemetryPoint);
    hipMemcpy(scan_buffer_d, scan_data, bytes, hipMemcpyHostToDevice);
    hipMemcpy(scan_size_d, &scan_size, sizeof(int), hipMemcpyHostToDevice);

    int dim = 1024;
    int n = dim * dim;
    LocalizedOrigin *result_d;
    LocalizedOrigin *result_h;
    checkCuda(hipMalloc((void **)&result_d, n*sizeof(LocalizedOrigin)));
    checkCuda(hipHostMalloc((void **)&result_h, n*sizeof(LocalizedOrigin)));

    if(!map_init){
        cudaInitMap<<<1, 1>>>(scan_buffer_d, scan_size_d, map_d, width_d, height_d);
        map_init = true;
    }
    cudaUpdateMap<<<dim, dim>>>(n, result_d, scan_buffer_d, scan_size_d, map_d, width_d, height_d);

    hipMemcpy(map_h, map_d, map_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(result_h, result_d, n*sizeof(LocalizedOrigin), hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++){
        if(result_h[i].score != 0){
            printf("Particle Filter Result: i[%d] x[%d] y[%d] angle[%.2f] score[%d]\n", i, result_h[i].x_offset, result_h[i].y_offset, result_h[i].angle_offset, result_h[i].score);
        }
    }

    hipHostFree(result_h);
    hipFree(result_d);

    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    float time;
    checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );

    printf("Map::update processed %d points and took %.2f ms\n", scan_size, time);

    CheckpointWriter::checkpoint("cuda", 2000,2000, scan_data, scan_size, map_h);

    return TelemetryPoint{0, 0, 0, 0, 0};
}