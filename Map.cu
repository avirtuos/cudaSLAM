#include "hip/hip_runtime.h"

#include "Map.h"

Map::Map(int width_arg, int height_arg, int scan_buffer_size)
{
    width = width_arg;
    height = height_arg;

    //Allocate scan buffer on the device as well as the size of scan
    const unsigned int scan_bytes = scan_buffer_size * sizeof(TelemetryPoint);
    checkCuda(hipMalloc((void **)&scan_buffer_d, scan_bytes));
    checkCuda(hipMalloc((void **)&scan_size_d, sizeof(int)));

    //Allocate pinned memory on the host and device for the current map
    map_bytes = width * height * sizeof(MapPoint);
    checkCuda(hipHostMalloc((void**)&map_h, map_bytes));
    checkCuda(hipMalloc((void**)&map_d, map_bytes));
    checkCuda(hipMalloc((void **)&width_d, sizeof(int)));
    checkCuda(hipMalloc((void **)&height_d, sizeof(int)));

    //copy the size of the map to the device
    //todo: in the future this should probably be auto-expandable based on the size of the mapped area
    //but evne using a static size is fine for area 30m x 30m which is more than enough for most hobby applications
    hipMemcpy(width_d, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(height_d, &height, sizeof(int), hipMemcpyHostToDevice);
}

Map::~Map()
{
    hipHostFree(map_h);
    hipFree(map_d);
    hipFree(width_d);
    hipFree(height_d);
    hipFree(scan_buffer_d);
    hipFree(scan_size_d);
}

__global__
void cudaUpdateMap(int *result, TelemetryPoint *scan_buffer, int *scan_size, MapPoint *map, int* map_width, int* map_height)
{
    *result=0;
    int offset = blockIdx.x*blockDim.x + threadIdx.x;
    printf("Hello from block %d, dim %d, thread %d, offset: %d\n", blockIdx.x, blockDim.x, threadIdx.x, offset);
    
    for(int i = 0; i < *scan_size || i < 10; i = i + (1 + offset)){
        TelemetryPoint *cur_point = scan_buffer+i;
        int pos = ((*map_height/2 + cur_point->y) * *map_width) + (*map_height/2 + cur_point->x);
        MapPoint *cur_map = map+pos;
        if(cur_map->occupancy < 1000) {
            cur_map->occupancy++;
        }
        *result = *result +1; 
    }
}

TelemetryPoint Map::update(int32_t search_distance, TelemetryPoint scan_data[], int scan_size)
{
    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );
    checkCuda( hipEventRecord(startEvent, 0) );

    printf("HERE1\n");
    const unsigned int bytes = scan_size * sizeof(TelemetryPoint);
    hipMemcpy(scan_buffer_d, scan_data, bytes, hipMemcpyHostToDevice);
    hipMemcpy(scan_size_d, &scan_size, sizeof(int), hipMemcpyHostToDevice);

    int *result_d;
    int *result_h;
    checkCuda(hipMalloc((void **)&result_d, sizeof(int)));
    checkCuda(hipHostMalloc((void **)&result_h, sizeof(int)));

printf("HERE2\n");
    cudaUpdateMap<<<1, 1>>>(result_d, scan_buffer_d, scan_size_d, map_d, width_d, height_d);
printf("HERE3\n");

    hipMemcpy(map_h, map_d, map_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(result_h, result_d, sizeof(int), hipMemcpyDeviceToHost);

    printf("Result: %d\n", *result_h);

    hipHostFree(result_h);
    hipFree(result_d);

    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    float time;
    checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );

    printf("Map::update processed %d points and took %.2f ms\n", scan_size, time);

    CheckpointWriter::checkpoint("cuda", 2000,2000, scan_data, scan_size, map_h);

    return TelemetryPoint{0, 0, 0, 0, 0};
}