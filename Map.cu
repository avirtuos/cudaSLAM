#include "hip/hip_runtime.h"

#include "Map.h"

Map::Map(int width_arg, int height_arg, int scan_buffer_size)
{
    map_init = false;
    width = width_arg;
    height = height_arg;

    //Allocate scan buffer on the device as well as the size of scan
    const unsigned int scan_bytes = scan_buffer_size * sizeof(TelemetryPoint);
    checkCuda(hipMalloc((void **)&scan_buffer_d, scan_bytes));
    checkCuda(hipMalloc((void **)&scan_size_d, sizeof(int)));

    //Allocate sim buffer on device
    int sim_size_h = 360 * scan_buffer_size;
    const unsigned int sim_bytes = sim_size_h * sizeof(SimTelemetryPoint);
    checkCuda(hipMalloc((void **)&sim_buffer_d, sim_bytes));
    checkCuda(hipMalloc((void **)&sim_size_d, sizeof(int)));
    hipMemcpy(sim_size_d, &sim_size_h, sizeof(int), hipMemcpyHostToDevice);

    //Allocate pinned memory on the host and device for the current map
    map_bytes = width * height * sizeof(MapPoint);
    checkCuda(hipHostMalloc((void **)&map_h, map_bytes));
    checkCuda(hipMalloc((void **)&map_d, map_bytes));
    checkCuda(hipMalloc((void **)&width_d, sizeof(int)));
    checkCuda(hipMalloc((void **)&height_d, sizeof(int)));

    //copy the size of the map to the device
    //todo: in the future this should probably be auto-expandable based on the size of the mapped area
    //but evne using a static size is fine for area 30m x 30m which is more than enough for most hobby applications
    hipMemcpy(width_d, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(height_d, &height, sizeof(int), hipMemcpyHostToDevice);

    map_update_dim = 512;
    const unsigned int n = map_update_dim * map_update_dim;
    checkCuda(hipMalloc((void **)&result_d, n * sizeof(LocalizedOrigin)));

    localized_size = 10000;
    checkCuda(hipHostMalloc((void **)&localized_result_h, localized_size * sizeof(LocalizedOrigin)));
    checkCuda(hipMalloc((void **)&localized_result_d, localized_size * sizeof(LocalizedOrigin)));
}

Map::~Map()
{
    hipHostFree(map_h);
    hipFree(map_d);
    hipFree(width_d);
    hipFree(height_d);
    hipFree(scan_buffer_d);
    hipFree(scan_size_d);
    hipFree(sim_buffer_d);
    hipFree(sim_size_d);
    hipFree(result_d);
    hipHostFree(localized_result_h);
    hipFree(localized_result_d);
}


__global__
void cudeGenerateParticleFilter(SimTelemetryPoint *sim_buffer, int *sim_size, TelemetryPoint *scan_buffer, int *scan_size)
{
    extern __shared__ TelemetryPoint scan_buffer_s[];

    for(int i = threadIdx.x; i < *scan_size; i += blockDim.x)
    {
        scan_buffer_s[i] = scan_buffer[i];
    }
    __syncthreads();

    int sim_num = blockIdx.x * blockDim.x + threadIdx.x;

    if(sim_num > *sim_size)
    {
        return;
    }

    int increment = gridDim.x * blockDim.x;
    for(int i = sim_num; i < 360 *  *scan_size; i += increment)
    {
        int scan_num = i % *scan_size;
        float distance = scan_buffer_s[scan_num].distance;
        float angle_num = scan_buffer_s[scan_num].angle + floorf(i / *scan_size);

        sim_buffer[i].x = roundf(__sinf (angle_num) * distance);
        sim_buffer[i].y = roundf(__cosf (angle_num) * distance);
    }
}



//TODO: this needs parrallism, like a map/reduce paradim. There were examples of this in the book where you use nested loops and sync threads.
__global__
void cudaLocalizeParticleFilter_slow(LocalizedOrigin *result, int result_size)
{
    LocalizedOrigin best;
    best.score = -1;
    for(int i = 0; i < result_size; i++)
    {
        if(result[i].score > best.score)
        {
            best.x_offset = result[i].x_offset;
            best.y_offset = result[i].y_offset;
            best.angle_offset = result[i].angle_offset;
            best.score = result[i].score;
        }
    }

    printf("BEST: x: %d  y: %d  a: %.2f  s: %d \n", best.x_offset, best.y_offset, best.angle_offset, best.score);
}

//TODO: this needs parrallism, like a map/reduce paradim. There were examples of this in the book where you use nested loops and sync threads.
__global__
void cudaLocalizeParticleFilter(LocalizedOrigin *output, int max_output_size, LocalizedOrigin *input, int input_size)
{
    extern __shared__ LocalizedOrigin localization[];

    if(blockDim.x >= max_output_size ||  gridDim.x >= max_output_size ){
        //kernel config exceeds buffer sizes
        if(blockIdx.x * blockDim.x + threadIdx.x == 0) {
            printf("Exiting due to insufficient buffer size");
        }
        return;
    }

    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    LocalizedOrigin best;
    best.score = -1;

    //do the first round.
    int increment = gridDim.x * blockDim.x;
    for(int i = offset; i < input_size; i += increment)
    {
        if(input[i].score > best.score)
        {
            best = input[i];
        }
    }

    localization[tid] = best;
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s && localization[tid].score < localization[tid+s].score) {
            localization[tid] = localization[tid+s];
        }
        __syncthreads();
    }

    if (tid == 0) output[blockIdx.x] = localization[0];
}


//TODO: There is opportunity to speed this up using hints from odometry or even just simple distance traveled estimates.
__global__
void cudaRunParticleFilter(int search_distance, LocalizedOrigin *result, SimTelemetryPoint *sim_buffer, int *sim_size, TelemetryPoint *scan_buffer, int *scan_size, MapPoint *map, int *map_width, int *map_height)
{
    extern __shared__ SimTelemetryPoint sim_buffer_s[];
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    int e_search_distance = search_distance / 2;
    int x_offset = (-1 * e_search_distance) + offset % (e_search_distance * 2);
    int y_offset = (-1 * e_search_distance) + floorf(offset / (e_search_distance * 2));

    long max_pos = *map_width * *map_height;

    int e_width = (*map_width / 2);
    int e_height = (*map_height / 2);

    LocalizedOrigin best;
    best.score = 0;

    //Try various angles - TODO: find better sampling technique here possibly even re-sampling
    for(int angle_offset = 0; angle_offset < 360; angle_offset++)
    {
        //For each point see if we have a hit
        int score = 0;
        int scan_point_offset = angle_offset * *scan_size;

        for(int i = threadIdx.x; i < *scan_size; i += blockDim.x)
        {
            sim_buffer_s[i] = sim_buffer[scan_point_offset + i];
        }

        __syncthreads();

        if(x_offset < search_distance && y_offset < search_distance)
        {
            for(int scan_point = 0; scan_point < *scan_size; scan_point++)
            {
                //SimTelemetryPoint *scan_point_ptr = sim_buffer_s + scan_point;
                int x = x_offset + sim_buffer_s[scan_point].x;
                int y = y_offset + sim_buffer_s[scan_point].y;

                int pos = ((e_height + y) * *map_width) + (e_width + x);

                if(pos < 0 || pos >= max_pos)
                {
                    continue;
                }

                //MapPoint *map_point = map + pos;
                score += map[pos].occupancy;
            }
        }

        if(best.score < score)
        {
            best.x_offset = x_offset;
            best.y_offset = y_offset;
            best.angle_offset = angle_offset;
            best.score = score;
        }

        __syncthreads();
    }

    result[offset].x_offset = best.x_offset;
    result[offset].y_offset = best.y_offset;
    result[offset].angle_offset = best.angle_offset;
    result[offset].score = best.score;
}


//TODO: So far we've only been working on Localization, we need to start thinking about mapping or rather
//when to update the map with newly scanned points. I suspect that cold start might be a special case but
//it needs more thinking. I like the idea of the map being fully mutable, not just additive which is what
//I've seen from other SLAM impls.
__global__
void cudaUpdateMap(TelemetryPoint *scan_buffer, int *scan_size, MapPoint *map, int *map_width, int *map_height)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    int map_size = *map_width * *map_height;
    for(int i = offset; i < map_size; i += gridDim.x * blockDim.x)
    {
        if(map[i].occupancy > 0)
        {
            //printf("MAP: i: %d, s: %d \n", map[i].occupancy);
            map[i].occupancy -= 1;
        }
    }

    //synchronizing across blocks would be better - TODO: get grid_group and sync working. Was facing linking errors with this.
    __syncthreads();

    if(offset >= *scan_size)
    {
        return;
    }

    TelemetryPoint *cur_point = scan_buffer + offset;
    int pos = ((*map_height / 2 + cur_point->y) * *map_width) + (*map_height / 2 + cur_point->x);

    if(pos > map_size)
    {
        return;
    }

    //printf("Point: x: %d y: %d, a: %.2f p: %d\n", cur_point->x, cur_point->y, cur_point->angle, pos);
    MapPoint *cur_map = map + pos;
    if(cur_map->occupancy < 225)
    {
        cur_map->occupancy += 25;
    }
    //printf("MAP: o: %d p: %d - x: %d. y: %d a:%.2f, q: %d\n", offset, pos, cur_point->x, cur_point->y, cur_point->angle, cur_point->quality);
}


TelemetryPoint Map::update(int32_t search_distance, TelemetryPoint scan_data[], int scan_size)
{

    hipProfilerStart();
    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );
    checkCuda( hipEventRecord(startEvent, 0) );

    const unsigned int bytes = scan_size * sizeof(TelemetryPoint);
    hipMemcpy(scan_buffer_d, scan_data, bytes, hipMemcpyHostToDevice);
    hipMemcpy(scan_size_d, &scan_size, sizeof(int), hipMemcpyHostToDevice);

    cudaUpdateMap <<< 32, 256 >>> (scan_buffer_d, scan_size_d, map_d, width_d, height_d);
    hipDeviceSynchronize();
    cudeGenerateParticleFilter <<< scan_size * 360 / 1024, 1024, scan_size *sizeof(TelemetryPoint) >>> (sim_buffer_d, sim_size_d, scan_buffer_d, scan_size_d);
    hipDeviceSynchronize();
    cudaRunParticleFilter <<< (search_distance*search_distance)/512 + 1, 512, scan_size *sizeof(SimTelemetryPoint)>>>(search_distance, result_d, sim_buffer_d, sim_size_d, scan_buffer_d, scan_size_d, map_d, width_d, height_d);
    hipDeviceSynchronize();
    //shared memory must be >= threads per block
    int num_localization_blocks = 32;
    cudaLocalizeParticleFilter <<< num_localization_blocks, 128, 128*sizeof(LocalizedOrigin)>>>(localized_result_d, localized_size, result_d, map_update_dim * map_update_dim);
    hipDeviceSynchronize();

    hipMemcpy(localized_result_h, localized_result_d, localized_size*sizeof(LocalizedOrigin), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    LocalizedOrigin best;
    best.score = -1;
    for(int i = 0; i < num_localization_blocks; i++){
        if(localized_result_h[i].score > best.score){
            best = localized_result_h[i];
        }
    }
    printf("BEST-FAST: x: %d  y: %d  a: %.2f  s: %d\n", best.x_offset, best.y_offset, best.angle_offset, best.score);

    checkCuda( hipMemcpy(map_h, map_d, map_bytes, hipMemcpyDeviceToHost));

    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    float time;
    checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );

    printf("Map::update processed %d points and took %.2f ms\n", scan_size, time);
    hipDeviceSynchronize();

    CheckpointWriter::checkpoint("cuda", width, height, scan_data, scan_size, map_h, &best);

    hipProfilerStop();
    return TelemetryPoint{0, 0, 0, 0, 0};
}